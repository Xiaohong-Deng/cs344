#include "hip/hip_runtime.h"
/* Udacity Homework 3
   HDR Tone-mapping

  Background HDR
  ==============

  A High Dynamic Range (HDR) image contains a wider variation of intensity
  and color than is allowed by the RGB format with 1 byte per channel that we
  have used in the previous assignment.  

  To store this extra information we use single precision floating point for
  each channel.  This allows for an extremely wide range of intensity values.

  In the image for this assignment, the inside of church with light coming in
  through stained glass windows, the raw input floating point values for the
  channels range from 0 to 275.  But the mean is .41 and 98% of the values are
  less than 3!  This means that certain areas (the windows) are extremely bright
  compared to everywhere else.  If we linearly map this [0-275] range into the
  [0-255] range that we have been using then most values will be mapped to zero!
  The only thing we will be able to see are the very brightest areas - the
  windows - everything else will appear pitch black.

  The problem is that although we have cameras capable of recording the wide
  range of intensity that exists in the real world our monitors are not capable
  of displaying them.  Our eyes are also quite capable of observing a much wider
  range of intensities than our image formats / monitors are capable of
  displaying.

  Tone-mapping is a process that transforms the intensities in the image so that
  the brightest values aren't nearly so far away from the mean.  That way when
  we transform the values into [0-255] we can actually see the entire image.
  There are many ways to perform this process and it is as much an art as a
  science - there is no single "right" answer.  In this homework we will
  implement one possible technique.

  Background Chrominance-Luminance
  ================================

  The RGB space that we have been using to represent images can be thought of as
  one possible set of axes spanning a three dimensional space of color.  We
  sometimes choose other axes to represent this space because they make certain
  operations more convenient.

  Another possible way of representing a color image is to separate the color
  information (chromaticity) from the brightness information.  There are
  multiple different methods for doing this - a common one during the analog
  television days was known as Chrominance-Luminance or YUV.

  We choose to represent the image in this way so that we can remap only the
  intensity channel and then recombine the new intensity values with the color
  information to form the final image.

  Old TV signals used to be transmitted in this way so that black & white
  televisions could display the luminance channel while color televisions would
  display all three of the channels.
  

  Tone-mapping
  ============

  In this assignment we are going to transform the luminance channel (actually
  the log of the luminance, but this is unimportant for the parts of the
  algorithm that you will be implementing) by compressing its range to [0, 1].
  To do this we need the cumulative distribution of the luminance values.

  Example
  -------

  input : [2 4 3 3 1 7 4 5 7 0 9 4 3 2]
  min / max / range: 0 / 9 / 9

  histo with 3 bins: [4 7 3]

  cdf : [4 11 14]


  Your task is to calculate this cumulative distribution by following these
  steps.

*/

#include "utils.h"

__global__
void shmem_reduce_kernel(float * d_max_out, float * d_min_out, const float * d_in) {
  
  assert(blockDim.x % 2 == 0);
  // according to stackoverflow, extern __shared__ array can only have one copy
  // in order to fulfill the need of 2 array, double the size
  extern __shared__ float max_min_data[];
  
  int index = threadIdx.x + blockIdx.x * blockDim.x;
  int idx_in_block = threadIdx.x;
  
  max_min_data[idx_in_block] = d_in[index];
  max_min_data[idx_in_block + blockDim.x] = d_in[index]
  __syncthreads();
  
  for (unsigned int s = blockDim.x / 2; s > 0; s >>= 1) {
    if (idx_in_block < s) {
      max_min_data[idx_in_block] = fmaxf(max_min_data[idx_in_block], max_min_data[idx_in_block + s]);
      min_data[idx_in_block + blockDim.x] = fminf(max_min_data[idx_in_block + blockDim.x], max_min_data[idx_in_block + blockDim.x + s]);
    }
    __syncthreads();
  }

  if (s == 0) {
    d_max_out[blockIdx.x] = max_data[0];
    d_min_out[blockIdx.x] = min_data[blockDim.x];
  }
}

__global__
void reduce(float * d_max_out,
            float * d_min_out
            float * d_max_intermediate,
            float * d_min_intermediate,
            float * d_in,
            int size) {
    const int maxThreadsPerBlock = 1024;
    int threads = maxThreadsPerBlock;
    // if size is not divisible by maxThreadPerBlock, do I need an extra block
    int blocks = size / maxThreadsPerBlock;
    shmem_reduce_kernel<<<blocks, threads, 2 * threads * sizeof(float)>>>
            (d_max_intermediate, d_min_intermediate, d_in);
    threads = blocks;
    blocks = 1;
    shmem_reduce_kernel<<<blocks, threads, 2 * threads * sizeof(float)>>>
            (d_max_out, d_min_out, d_intermediate);
}

__global__
void find_max_and_min(float * d_in, float * global_max, float * global_min,
                      size_t numRows, size_t numCols) {
  const int ARRAY_SIZE = numRows * numCols;
  const int ARRAY_BYTES = ARRAY_SIZE * sizeof(float);
  float *d_max_intermediate, *d_min_intermediate;

  checkCudaErrors(hipMalloc((void **) &d_max_intermediate, ARRAY_BYTES));
  checkCudaErrors(hipMalloc((void **) &d_min_intermediate, ARRAY_BYTES));

  reduce(global_max, global_min, d_max_intermediate, d_min_intermediate, d_in, ARRAY_SIZE);

  checkCudaErrors(hipFree(d_max_intermediate));
  checkCudaErrors(hipFree(d_min_intermediate));
}

void scatter_kernel(const float * d_in, const size_t numBins, const float lumRange, int *d_bins) {

}

void your_histogram_and_prefixsum(const float* const d_logLuminance,
                                  unsigned int* const d_cdf,
                                  float &min_logLum,
                                  float &max_logLum,
                                  const size_t numRows,
                                  const size_t numCols,
                                  const size_t numBins)
{
  //TODO
  /*Here are the steps you need to implement
    1) find the minimum and maximum value in the input logLuminance channel
       store in min_logLum and max_logLum
    2) subtract them to find the range
    3) generate a histogram of all the values in the logLuminance channel using
       the formula: bin = (lum[i] - lumMin) / lumRange * numBins
    4) Perform an exclusive scan (prefix sum) on the histogram to get
       the cumulative distribution of luminance values (this should go in the
       incoming d_cdf pointer which already has been allocated for you)       */
  find_max_and_min(d_logLuminance, &max_logLum, &min_logLum, numRows, numCols);

  float lumRange = max_logLum - min_logLum;

}
